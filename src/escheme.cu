#include "hip/hip_runtime.h"
/*
The MIT License (MIT)

Copyright (c) 2014 Leonardo Kewitz

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "./cuda_snippets.h"
#include "./escheme.h"

// Kernel de pré-processamento responsável por calcular as matrizes de contribu-
// ição de todos os elementos.
//    ne: número de elementos.
//    elements: array de elementos da malha.
//    elements: array de nós da malha.
__global__ void kernel_integration(int ne, elementri *elements, node *nodes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    elementri E = elements[i];
    node N1 = nodes[E.nodes[0]], N2 = nodes[E.nodes[1]], N3 = nodes[E.nodes[2]];

    // Calcula argumentos necessários
    float J1, J2, J3, J4, dJ;
    J1 = N2.x - N1.x;
    J2 = N2.y - N1.y;
    J3 = N3.x - N1.x;
    J4 = N3.y - N1.y;
    dJ = 2*(J1*J4 - J3*J2);

    // Calcula a matriz de contribuições do elemento.
    elements[i].matriz[0] = dJ != 0.0 ?
        (powf(J2-J4, 2.0) + powf(J3-J1, 2.0))/dJ : 0.0;
    elements[i].matriz[1] = dJ != 0.0 ?
        (powf(J4, 2.0) + powf(J3, 2.0))/dJ : 0.0;
    elements[i].matriz[2] = dJ != 0.0 ?
        (powf(J2, 2.0) + powf(J1, 2.0))/dJ : 0.0;
    elements[i].matriz[3] = dJ != 0.0 ?
        ((J2-J4)*J4 - (J3-J1)*J3)/dJ : 0.0;
    elements[i].matriz[4] = dJ != 0.0 ?
        ((J3-J1)*J1 - (J2-J4)*J2)/dJ : 0.0;
    elements[i].matriz[5] = dJ != 0.0 ?
        (J4*-1*J2 - J3*J1)/dJ : 0.0;
}

// Kernel de pré-processamento responsável por calcular diag_sum e right_sum.
//    ne: número de elementos.
//    elements: array de elementos da malha.
//    V: vetor de tensões dos nós.
//    dsum: vetor diag_sum.
//    rsum: vetor right_sum.
__global__ void kernel_preprocess(int ne, elementri * elements, float * V,
                                  float * dsum, float * rsum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    int n1, n2, n3;
    elementri E = elements[i];
    n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

    atomicAdd(&dsum[n1], E.matriz[0]);
    atomicAdd(&dsum[n2], E.matriz[1]);
    atomicAdd(&dsum[n3], E.matriz[2]);

    atomicAdd(&rsum[n1], -E.matriz[3]*V[n2] -E.matriz[4]*V[n3]);
    atomicAdd(&rsum[n2], -E.matriz[3]*V[n1] -E.matriz[5]*V[n3]);
    atomicAdd(&rsum[n3], -E.matriz[4]*V[n1] -E.matriz[5]*V[n2]);
}

// Kernel de pré-condicionamento.
__global__ void kernel_precond(int nn, node * nodes, float * dsum, float * rsum,
                               float * R, float * P, float * V) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nn) return;

    float ri = nodes[i].calc ? rsum[i] - dsum[i]*V[i] : 0.0;
    R[i] = ri;
    P[i] = ri;
}

__global__ void kernel_iter_element(int ne, elementri * elements, float * u,
                                    float * p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    elementri E = elements[i];
    int n1 = E.nodes[0], n2 = E.nodes[1], n3 = E.nodes[2];

    u[n1] += E.matriz[0]*p[n1] + E.matriz[3]*p[n2] + E.matriz[4]*p[n3];
    u[n2] += E.matriz[3]*p[n1] + E.matriz[1]*p[n2] + E.matriz[5]*p[n3];
    u[n3] += E.matriz[4]*p[n1] + E.matriz[5]*p[n2] + E.matriz[2]*p[n3];
}

__global__ void kernel_util_zero(int size, float * vec) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    vec[i] = 0.0;
}

// Função externa que processa o problema na GPU.
//    ne: número de elementos.
//    nn: número de nós.
//    kmax: número máximo de iterações.
//    errmin: erro mínimo para considerar a convergência do resultado.
//    elements: array de elementos da malha.
//    nodes: array de nós da malha.
//    V: vetor de tensões dos nós.
//    verbose: se 'true' imprime informações do algorítmo.
//    bench: array de tempos de processamento para benchmarking.
extern "C" int runGPU(int ne, int nn, int kmax, float errmin,
                      elementri *elements, node *nodes, float *V, bool verbose,
                      float *bench) {
    int i, k = 1;
    const dim3 threads(512);
    const dim3 elementblocks(1 + ne/512);
    const dim3 nodeblocks(1 + nn/512);

    // Array Sizes
    size_t s_Elements = sizeof(elementri)*ne,
           s_Nodes = sizeof(node)*nn,
           s_V = sizeof(float)*nn;

    // Device Arrays.
    float *_dsum, *_rsum, *_V, *_U, *_P, *_R;
    node *_nodes;
    elementri *_elements;
    CudaSafeCall(hipMalloc(&_elements, s_Elements));
    CudaSafeCall(hipMalloc(&_nodes, s_Nodes));
    CudaSafeCall(hipMalloc(&_V, s_V));
    CudaSafeCall(hipMalloc(&_U, s_V));
    CudaSafeCall(hipMalloc(&_P, s_V));
    CudaSafeCall(hipMalloc(&_R, s_V));
    CudaSafeCall(hipMalloc(&_dsum, s_V));
    CudaSafeCall(hipMalloc(&_rsum, s_V));

    CudaSafeCall(hipMemcpy(_elements, elements, s_Elements, hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(_V, V, s_V, hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(_nodes, nodes, s_Nodes, hipMemcpyHostToDevice));

    kernel_integration<<<elementblocks, threads>>>(ne, _elements, _nodes);
    kernel_util_zero<<<nodeblocks, threads>>>(nn, _dsum);
    kernel_util_zero<<<nodeblocks, threads>>>(nn, _rsum);
    hipDeviceSynchronize();
    kernel_preprocess<<<elementblocks, threads>>>(ne, _elements, _V, _dsum, _rsum);
    hipDeviceSynchronize();
    kernel_precond<<<nodeblocks, threads>>>(nn, _nodes, _dsum, _nsum, _R, _P, _V);

    return k;
}

// Função externa que processa o problema no CPU.
//    ne: número de elementos.
//    nn: número de nós.
//    kmax: número máximo de iterações.
//    errmin: erro mínimo para considerar a convergência do resultado.
//    elements: array de elementos da malha.
//    nodes: array de nós da malha.
//    V: vetor de tensões dos nós.
//    verbose: se 'true' imprime informações do algorítmo.
//    bench: array de tempos de processamento para benchmarking.
extern "C" int runCPU(int ne, int nn, int kmax, float errmin,
                      elementri *elements, node *nodes, float *V, bool verbose,
                      float *bench) {
    int i, k;

    // Pre-processamento. Calcula as matrizes de contribuição dos elementos.
    float J1, J2, J3, J4, dJ;
    elementri E;
    node N1, N2, N3;
    for (i = 0; i < ne; i++) {
        E = elements[i];
        N1 = nodes[E.nodes[0]]; N2 = nodes[E.nodes[1]]; N3 = nodes[E.nodes[2]];

        // Calcula argumentos necessários
        J1 = N2.x - N1.x;
        J2 = N2.y - N1.y;
        J3 = N3.x - N1.x;
        J4 = N3.y - N1.y;
        dJ = 2*(J1*J4 - J3*J2);

        // Calcula a matriz de contribuições do elemento.
        elements[i].matriz[0] = dJ != 0.0 ?
            (pow(J2-J4, 2.0) + pow(J3-J1, 2.0))/dJ : 0.0;
        elements[i].matriz[1] = dJ != 0.0 ?
            (pow(J4, 2.0) + pow(J3, 2.0))/dJ : 0.0;
        elements[i].matriz[2] = dJ != 0.0 ?
            (pow(J2, 2.0) + pow(J1, 2.0))/dJ : 0.0;
        elements[i].matriz[3] = dJ != 0.0 ?
            ((J2-J4)*J4 - (J3-J1)*J3)/dJ : 0.0;
        elements[i].matriz[4] = dJ != 0.0 ?
            ((J3-J1)*J1 - (J2-J4)*J2)/dJ : 0.0;
        elements[i].matriz[5] = dJ != 0.0 ?
            (J4*-1*J2 - J3*J1)/dJ : 0.0;
    }

    // Pre-processamento. Calcula dsum e rsum.
    int n1, n2, n3;
    float *rsum = (float*)malloc(nn*sizeof(float));
    float *dsum = (float*)malloc(nn*sizeof(float));
    // Inicialização dos vetores.
    for (i = 0; i < nn; i++) {
        rsum[i] = 0.0;
        dsum[i] = 0.0;
    }
    for (i = 0; i < ne; i++) {
        E = elements[i];
        n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

        dsum[n1] += E.matriz[0];
        dsum[n2] += E.matriz[1];
        dsum[n3] += E.matriz[2];

        rsum[n1] += - E.matriz[3]*V[n2] - E.matriz[4]*V[n3];
        rsum[n2] += - E.matriz[3]*V[n1] - E.matriz[5]*V[n3];
        rsum[n3] += - E.matriz[4]*V[n1] - E.matriz[5]*V[n2];
    }

    // CG
    float ri, alpha, beta, sum1, sum2, sum3 = 1, sum4;
    float *r = (float*)malloc(nn*sizeof(float));
    float *p = (float*)malloc(nn*sizeof(float));
    float *u = (float*)malloc(nn*sizeof(float));

    // Pré-condicionamento.
    for (i = 0; i < nn; i++) {
        ri = nodes[i].calc ? rsum[i] - dsum[i]*V[i] : 0.0;
        p[i] = ri;
        r[i] = ri;
    }

    k = 1;
    while (k < kmax && fabs(sqrt(sum3)) > errmin) {
        for (i = 0; i < nn; i++)
            u[i] = 0.0;

        for (i = 0; i < ne; i++) {
            E = elements[i];
            n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

            u[n1] += E.matriz[0]*p[n1] + E.matriz[3]*p[n2] + E.matriz[4]*p[n3];
            u[n2] += E.matriz[3]*p[n1] + E.matriz[1]*p[n2] + E.matriz[5]*p[n3];
            u[n3] += E.matriz[4]*p[n1] + E.matriz[5]*p[n2] + E.matriz[2]*p[n3];
        }

        for (i = 0; i < nn; i++)
            if (!nodes[i].calc)
                u[i] = p[i];

        sum1 = 0.0; sum2 = 0.0;
        for (i = 0; i < nn; i++) {
            sum1 += p[i]*r[i];
            sum2 += p[i]*u[i];
        }

        alpha = sum2 != 0.0 ? sum1/sum2 : 0.0;
        for (i = 0; i < nn; i++) {
            V[i] += alpha*p[i];
            r[i] -= alpha*u[i];
        }

        sum3 = 0.0; sum4 = 0.0;
        for (i = 0; i < nn; i++) {
            sum3 += r[i]*r[i];
            sum4 += r[i]*u[i];
        }

        beta = sum2 != 0.0 ? -sum4/sum2 : 0.0;
        for (i = 0; i < nn; i++) {
            p[i] = r[i] + beta*p[i];
        }

        k++;
    }

    free(dsum);
    free(rsum);
    free(r);
    free(p);
    free(u);

    return k;
}

// Sadiku's Numerical Techniques in Electromagnetics. pg.712
extern "C" int testeCG(int n, int kmax, float err, float* A, float* x,
                       float* b) {
    int i, j, k = 1;
    float alpha, beta, sum1, sum2, sum3 = 1, sum4;
    float *r = (float*)malloc(n*sizeof(float));
    float *p = (float*)malloc(n*sizeof(float));
    float *u = (float*)malloc(n*sizeof(float));

    for (i = 0; i < n; i++) {
        p[i] = b[i];
        r[i] = b[i];
    }

    while (k < kmax && fabs(sqrt(sum3)) > err) {
        for (j = 0; j < n; j++) {
            u[j] = 0.0;
            for (i = 0; i < n; i++)
                u[j] += A[i*n + j]*p[i];
        }

        sum1 = 0.0; sum2 = 0.0;
        for (i = 0; i < n; i++) {
            sum1 += p[i]*r[i];
            sum2 += p[i]*u[i];
        }

        alpha = sum2 != 0.0 ? sum1/sum2 : 0.0;

        for (i = 0; i < n; i++) {
            x[i] += alpha*p[i];
            r[i] -= alpha*u[i];
        }

        sum3 = 0.0; sum4 = 0.0;
        for (i = 0; i < n; i++) {
            sum3 += r[i]*r[i];
            sum4 += r[i]*u[i];
        }

        beta = sum2 != 0.0 ? -sum4/sum2 : 0.0;

        for (i = 0; i < n; i++) {
            p[i] = r[i] + beta*p[i];
        }

        k++;
    }

    free(r);
    free(p);
    free(u);

    return k;
}
