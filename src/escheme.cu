#include "hip/hip_runtime.h"
/*
 The MIT License (MIT)

 Copyright (c) 2014 Leonardo Kewitz

 Permission is hereby granted, free of charge, to any person obtaining a copy
 of this software and associated documentation files (the "Software"), to deal
 in the Software without restriction, including without limitation the rights
 to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 copies of the Software, and to permit persons to whom the Software is
 furnished to do so, subject to the following conditions:

 The above copyright notice and this permission notice shall be included in all
 copies or substantial portions of the Software.

 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 SOFTWARE.
 */

#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "./cuda_snippets.h"
#include "./escheme.h"

#define BSIZE 512
#define putf(a, b) smemcpy(a, b, sizeof(float), hipMemcpyHostToDevice);
#define getf(a, b) smemcpy(a, b, sizeof(float), hipMemcpyDeviceToHost);
#define CUDA true

extern "C" unsigned int alloc(const int nn) {
    hipDeviceProp_t prop = getInfo();
    unsigned int gm = prop.totalGlobalMem*.8 - sizeof(float)*nn*6
                      - sizeof(float)*4;
    hipDeviceReset();
    return cast(unsigned int, gm / (sizeof(node) + 6*sizeof(element)));
}

#if CUDA
// Kernel de responsável por calcular as matrizes de contribuição de um
// elemento.
__global__ void kernel_element(int ne, element *elements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    element E = elements[i];

    // Calcula gradN
    float q1 = E.y[1]-E.y[2], q2 = E.y[2]-E.y[0], q3 = E.y[0]-E.y[1];
    float r1 = E.x[2]-E.x[1], r2 = E.x[0]-E.x[2], r3 = E.x[1]-E.x[0];
    // Calcula det(gradN)
    float det = E.x[1]*E.y[2] + E.x[0]*E.y[1] + E.x[2]*E.y[0]
              - E.x[0]*E.y[2] - E.x[2]*E.y[1] - E.x[1]*E.y[0];
    float cof = (E.mat/det)/2;
    // Calcula a matriz de contribuições do elemento.
    elements[i].matriz[0] = (q1*q1 + r1*r1)*cof;
    elements[i].matriz[1] = (q2*q2 + r2*r2)*cof;
    elements[i].matriz[2] = (q3*q3 + r3*r3)*cof;
    elements[i].matriz[3] = (q1*q2 + r1*r2)*cof;
    elements[i].matriz[4] = (q1*q3 + r1*r3)*cof;
    elements[i].matriz[5] = (q2*q3 + r2*r3)*cof;
}

// Kernel de pré-processamento responsável por calcular diag_sum e right_sum.
__global__ void kernel_preprocess(int ne, element *elements, float *V,
    float *dsum, float *rsum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    int n1, n2, n3;
    element E = elements[i];
    n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

    atomicAdd(&dsum[n1], E.matriz[0]);
    atomicAdd(&dsum[n2], E.matriz[1]);
    atomicAdd(&dsum[n3], E.matriz[2]);

    atomicAdd(&rsum[n1], - E.matriz[3]*V[n2] - E.matriz[4]*V[n3]);
    atomicAdd(&rsum[n2], - E.matriz[3]*V[n1] - E.matriz[5]*V[n3]);
    atomicAdd(&rsum[n3], - E.matriz[4]*V[n1] - E.matriz[5]*V[n2]);
}

// Kernel de pré-condicionamento.
__global__ void kernel_precond(int nn, node *nodes, float *dsum, float *rsum,
    float *R, float *P, float *V) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nn) return;

    node N = nodes[i];

    float ri = N.calc ? rsum[N.i] - dsum[N.i]*V[N.i] : 0.0;
    R[N.i] = ri;
    P[N.i] = ri;
}

// U = SS*P
__global__ void kernel_iter_element(int ne, element *elements, float *U,
    float *P) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    element E = elements[i];
    int n1 = E.nodes[0], n2 = E.nodes[1], n3 = E.nodes[2];

    atomicAdd(&U[n1], E.matriz[0]*P[n1] + E.matriz[3]*P[n2] +
              E.matriz[4]*P[n3]);
    atomicAdd(&U[n2], E.matriz[3]*P[n1] + E.matriz[1]*P[n2] +
              E.matriz[5]*P[n3]);
    atomicAdd(&U[n3], E.matriz[4]*P[n1] + E.matriz[5]*P[n2] +
              E.matriz[2]*P[n3]);
}

// Corrige os valores de U para nós submetidos à condição de contorno.
__global__ void kernel_iter_element_fix(int nn, node *nodes, float *U,
    float *P) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= nn) return;

    node N = nodes[i];
    if (!N.calc) {
        U[N.i] = P[N.i];
    }
}

// vec[i] = 0.0f
__global__ void kernel_util_zero(int size, float *vec) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    vec[i] = 0.0;
}

// sum += vecA[i]*vecB[i]
__global__ void kernel_util_vecsummult(int size, float *vecA, float *vecB,
    float *sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int ti = threadIdx.x;
    __shared__ float _sum[BSIZE];


    _sum[ti] = (i < size) ? vecA[i]*vecB[i] : 0.0f;
    __syncthreads();

    for (int s = blockDim.x/2; s > 0; s >>= 1) {
       if (ti < s)
           _sum[ti] += _sum[ti + s];

       __syncthreads();
    }
    if (ti == 0) {
        atomicAdd(sum, _sum[0]);
    }
}

// vecA[i] += scalar * vecB[i]
__global__ void kernel_util_addtovec(int size, const float scalar, float *vecA,
    float *vecB) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    vecA[i] += scalar*vecB[i];
}

// vecA[i] = vecB[i] + scalar*vecC[i]
__global__ void kernel_util_addtovec2(int size, const float scalar, float *vecA,
    float *vecB, float *vecC) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    vecA[i] = vecB[i] + scalar*vecC[i];
}

// Função externa que processa o problema na GPU.
extern "C" int runGPU(int ng, int nn, int kmax, float errmin, group *groups,
    float *V, bool verbose, float *bench) {
    clock_t t = clock();
    int i, k = 1;
    unsigned int maxn = alloc(nn);

    // Array Sizes
    size_t s_Elements = sizeof(element)*maxn*6,
           s_Nodes = sizeof(node)*maxn,
           s_V = sizeof(float)*nn;

    // Scalars.
    float sum1 = 0.0f, sum2 = 0.0f, sum3 = 1.0f, sum4 = 0.0f, alpha = 0.0f,
          beta = 0.0f, *_sum1, *_sum2, *_sum3, *_sum4;

    // Device Arrays.
    float *_dsum, *_rsum, *_V, *_U, *_P, *_R;
    group *G;
    node *_nodes;
    element *_elements;
    smalloc(&_dsum, s_V); smalloc(&_rsum, s_V);
    smalloc(&_elements, s_Elements); smalloc(&_nodes, s_Nodes);
    smalloc(&_sum1, sizeof(float)); smalloc(&_sum2, sizeof(float));
    smalloc(&_sum3, sizeof(float)); smalloc(&_sum4, sizeof(float));
    smalloc(&_V, s_V); smalloc(&_U, s_V); smalloc(&_P, s_V); smalloc(&_R, s_V);

    smemcpy(_V, V, s_V, hipMemcpyHostToDevice);

    kernel_util_zero<<<(1+nn/BSIZE), BSIZE>>>(nn, _dsum);
    kernel_util_zero<<<(1+nn/BSIZE), BSIZE>>>(nn, _rsum);
    hipDeviceSynchronize();
    for (i = 0; i < ng; i++) {
        G = &groups[i];
        smemcpy(_elements, G->elements, sizeof(element)*G->ne,
            hipMemcpyHostToDevice);
        kernel_element<<<(1+G->ne/BSIZE), BSIZE>>>(G->ne, _elements);
        hipDeviceSynchronize();
        kernel_preprocess<<<(1+G->ne/BSIZE), BSIZE>>>(G->ne, _elements, _V,
            _dsum, _rsum);
        hipDeviceSynchronize();
    }
    for (i = 0; i < ng; i++) {
        G = &groups[i];
        smemcpy(_nodes, G->nodes, sizeof(node)*G->nn, hipMemcpyHostToDevice);
        kernel_precond<<<(1+G->nn/BSIZE), BSIZE>>>(G->nn, _nodes, _dsum, _rsum,
            _R, _P, _V);
        hipDeviceSynchronize();
    }

    while (k < kmax && fabs(sqrt(sum3)) > errmin) {
        // U[] = 0
        kernel_util_zero<<<(1+nn/BSIZE), BSIZE>>>(nn, _U);
        hipDeviceSynchronize();
        for (i = 0; i < ng; i++) {
            G = &groups[i];
            if (ng > 1)
                smemcpy(_elements, G->elements, sizeof(element)*G->ne,
                    hipMemcpyHostToDevice);
            kernel_element<<<(1+G->ne/BSIZE), BSIZE>>>(G->ne, _elements);
            hipDeviceSynchronize();
            kernel_iter_element<<<(1+G->ne/BSIZE), BSIZE>>>(G->ne, _elements,
                _U, _P);
            hipDeviceSynchronize();
        }
        for (i = 0; i < ng; i++) {
            G = &groups[i];
            if (ng > 1)
                smemcpy(_nodes, G->nodes, sizeof(node)*G->nn,
                    hipMemcpyHostToDevice);
            kernel_iter_element_fix<<<(1+G->nn/BSIZE), BSIZE>>>(G->nn, _nodes,
                _U, _P);
            hipDeviceSynchronize();
        }

        sum1 = 0.0f; sum2 = 0.0f;
        putf(_sum1, &sum1); putf(_sum2, &sum2);
        kernel_util_vecsummult<<<(1+nn/BSIZE), BSIZE>>>(nn, _P, _R, _sum1);
        kernel_util_vecsummult<<<(1+nn/BSIZE), BSIZE>>>(nn, _P, _U, _sum2);
        hipDeviceSynchronize();
        getf(&sum1, _sum1); getf(&sum2, _sum2);

        alpha = sum2 != 0.0 ? sum1/sum2 : 0.0;
        kernel_util_addtovec<<<(1+nn/BSIZE), BSIZE>>>(nn, alpha, _V, _P);
        kernel_util_addtovec<<<(1+nn/BSIZE), BSIZE>>>(nn, -alpha, _R, _U);
        hipDeviceSynchronize();

        sum3 = 0.0f; sum4 = 0.0f;
        putf(_sum3, &sum3); putf(_sum4, &sum4);
        kernel_util_vecsummult<<<(1+nn/BSIZE), BSIZE>>>(nn, _R, _R, _sum3);
        kernel_util_vecsummult<<<(1+nn/BSIZE), BSIZE>>>(nn, _R, _U, _sum4);
        hipDeviceSynchronize();
        getf(&sum3, _sum3); getf(&sum4, _sum4);

        beta = sum2 != 0.0 ? -sum4/sum2 : 0.0;
        kernel_util_addtovec2<<<(1+nn/BSIZE), BSIZE>>>(nn, beta, _P, _R, _P);
        hipDeviceSynchronize();

        k++;
    }

    smemcpy(V, _V, s_V, hipMemcpyDeviceToHost);

    hipFree(_V); hipFree(_U); hipFree(_P); hipFree(_R);
    hipFree(_sum1); hipFree(_sum2); hipFree(_sum3); hipFree(_sum4);
    hipFree(_elements); hipFree(_nodes);
    hipFree(_dsum); hipFree(_rsum);

    t = clock() - t;
    bench[0] = cast(float, t)/CLOCKS_PER_SEC;
    return k;
}
#endif

void integ_element(element *E) {
    float mat = E->mat;
    // Calcula gradN
    float q1 = E->y[1]-E->y[2], q2 = E->y[2]-E->y[0], q3 = E->y[0]-E->y[1];
    float r1 = E->x[2]-E->x[1], r2 = E->x[0]-E->x[2], r3 = E->x[1]-E->x[0];
    // Calcula det(gradN)
    float det = E->x[1]*E->y[2] + E->x[0]*E->y[1] + E->x[2]*E->y[0]
              - E->x[0]*E->y[2] - E->x[2]*E->y[1] - E->x[1]*E->y[0];
    float cof = (mat/det)/2.0;
    assert(!isnan(det));
    assert(!isnan(cof));
    // Calcula a matriz de contribuições do elemento.
    E->matriz[0] = (powf(q1, 2.0) + powf(r1, 2.0))*cof;
    E->matriz[1] = (powf(q2, 2.0) + powf(r2, 2.0))*cof;
    E->matriz[2] = (powf(q3, 2.0) + powf(r3, 2.0))*cof;
    E->matriz[3] = (q1*q2 + r1*r2)*cof;
    E->matriz[4] = (q1*q3 + r1*r3)*cof;
    E->matriz[5] = (q2*q3 + r2*r3)*cof;
}
// Função externa que processa o problema no CPU.
extern "C" int runCPU(int ng, int nn, int kmax, float errmin, group *groups,
    float *V, bool verbose, float *bench) {
    clock_t t = clock();
    unsigned int i, j, k;
    element *E;
    group *G;
    node N;

    // Pre-processamento. Calcula dsum e rsum.
    int n1, n2, n3;
    float *rsum = cast(float*, malloc(nn*sizeof(float)));
    float *dsum = cast(float*, malloc(nn*sizeof(float)));
    // Inicialização dos vetores.
    for (i = 0; i < nn; i++) {
        rsum[i] = 0.0f;
        dsum[i] = 0.0f;
    }
    for (i = 0; i < ng; i++) {
        G = &groups[i];
        for (j = 0; j < G->ne; j++) {
            E = &G->elements[j];
            integ_element(E);

            n1 = E->nodes[0]; n2 = E->nodes[1]; n3 = E->nodes[2];

            dsum[n1] += E->matriz[0];
            dsum[n2] += E->matriz[1];
            dsum[n3] += E->matriz[2];

            rsum[n1] += -E->matriz[3]*V[n2] -E->matriz[4]*V[n3];
            rsum[n2] += -E->matriz[3]*V[n1] -E->matriz[5]*V[n3];
            rsum[n3] += -E->matriz[4]*V[n1] -E->matriz[5]*V[n2];
        }
    }

    // CG
    float ri, alpha, beta, sum1, sum2, sum3 = 1.0f, sum4;
    float *r = cast(float*, malloc(nn*sizeof(float)));
    float *p = cast(float*, malloc(nn*sizeof(float)));
    float *u = cast(float*, malloc(nn*sizeof(float)));

    // Pré-condicionamento.
    for (i = 0; i < ng; i++) {
        G = &groups[i];
        for (j = 0; j < G->nn; j++) {
            N = G->nodes[j];
            ri = N.calc ? rsum[N.i] - dsum[N.i]*V[N.i] : 0.0f;
            r[N.i] = ri;
            p[N.i] = ri;
        }
    }

    k = 1;
    while (k < kmax && fabs(sqrt(sum3)) > errmin) {
        for (i = 0; i < nn; i++) {
            u[i] = 0.0;
        }

        for (i = 0; i < ng; i++) {
            G = &groups[i];
            for (j = 0; j < G->ne; j++) {
                E = &G->elements[j];
                integ_element(E);

                n1 = E->nodes[0]; n2 = E->nodes[1]; n3 = E->nodes[2];
                u[n1] += E->matriz[0]*p[n1] + E->matriz[3]*p[n2]
                         + E->matriz[4]*p[n3];
                u[n2] += E->matriz[3]*p[n1] + E->matriz[1]*p[n2]
                         + E->matriz[5]*p[n3];
                u[n3] += E->matriz[4]*p[n1] + E->matriz[5]*p[n2]
                         + E->matriz[2]*p[n3];
            }
            for (j = 0; j < G->nn; j++) {
                N = G->nodes[j];
                if (!N.calc)
                    u[N.i] = p[N.i];
            }
        }

        sum1 = 0.0; sum2 = 0.0;
        for (i = 0; i < nn; i++) {
            sum1 += p[i]*r[i];
            sum2 += p[i]*u[i];
        }

        alpha = sum2 != 0.0 ? sum1/sum2 : 0.0;
        for (i = 0; i < nn; i++) {
            V[i] += alpha*p[i];
            r[i] -= alpha*u[i];
        }

        sum3 = 0.0; sum4 = 0.0;
        for (i = 0; i < nn; i++) {
            sum3 += r[i]*r[i];
            sum4 += r[i]*u[i];
        }

        beta = sum2 != 0.0 ? -sum4/sum2 : 0.0f;
        for (i = 0; i < nn; i++) {
            p[i] = r[i] + beta*p[i];
        }

        k++;
    }

    free(dsum);
    free(rsum);
    free(r);
    free(p);
    free(u);

    t = clock() - t;
    bench[0] = cast(float, t)/CLOCKS_PER_SEC;
    return k;
}
