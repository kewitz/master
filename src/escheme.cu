#include "hip/hip_runtime.h"
/*
 The MIT License (MIT)

 Copyright (c) 2014 Leonardo Kewitz

 Permission is hereby granted, free of charge, to any person obtaining a copy
 of this software and associated documentation files (the "Software"), to deal
 in the Software without restriction, including without limitation the rights
 to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 copies of the Software, and to permit persons to whom the Software is
 furnished to do so, subject to the following conditions:

 The above copyright notice and this permission notice shall be included in all
 copies or substantial portions of the Software.

 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 SOFTWARE.
 */

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_snippets.h"
#include "escheme.h"


// Função externa que processa o problema no CPU.
//    ne: número de elementos.
//    nn: número de nós.
//    kmax: número máximo de iterações.
//    errmin: erro mínimo para considerar a convergência do resultado.
//    elements: array de elementos da malha.
//    nodes: array de nós da malha.
//    V: vetor de tensões dos nós.
//    verbose: se 'true' imprime informações do algorítmo.
//    bench: array de tempos de processamento para benchmarking.
extern "C" int runCPU(int ne, int nn, int kmax, float errmin,
                      elementri *elements, node *nodes, float *V, bool verbose,
                      float *bench) {
    int i;
    // clock_t t;

    float *rsum = (float*) malloc(nn*sizeof(float));
    float *dsum = (float*) malloc(nn*sizeof(float));
    float *r = (float*) malloc(nn*sizeof(float));
    float *z = (float*) malloc(nn*sizeof(float));
    float *p = (float*) malloc(nn*sizeof(float));
    float *q = (float*) malloc(nn*sizeof(float));
    // float *Vos = (float*) malloc(nn*sizeof(float));
    // memcpy(Vos, V, nn*sizeof(float));

    // Inicialização dos vetores.
    for (i = 0; i < nn; i++) {
        rsum[i] = 0.0;
        dsum[i] = 0.0;
    }

    // Pre-processamento. Calcula as matrizes de contribuição dos elementos.
    float J1, J2, J3, J4, dJ;
    elementri E;
    node N1, N2, N3;
    for (i = 0; i < ne; i++) {
        E = elements[i];
        N1 = nodes[E.nodes[0]]; N2 = nodes[E.nodes[1]]; N3 = nodes[E.nodes[2]];

        // Calcula argumentos necessários
        J1 = N2.x - N1.x;
        J2 = N2.y - N1.y;
        J3 = N3.x - N1.x;
        J4 = N3.y - N1.y;
        dJ = 2*(J1*J4 - J3*J2);

        // Calcula a matriz de contribuições do elemento.
        elements[i].matriz[0] = dJ != 0 ? (pow(J2-J4,2) + pow(J3-J1,2))*E.eps/dJ : 0.0;         // C11
        elements[i].matriz[1] = dJ != 0 ? (pow(J4,2) + pow(J3,2))*E.eps/dJ : 0.0;               // C22
        elements[i].matriz[2] = dJ != 0 ? (pow(J2,2) + pow(J1,2))*E.eps/dJ : 0.0;               // C33
        elements[i].matriz[3] = dJ != 0 ? ((J2-J4)*J4 - (J3-J1)*J3)*E.eps/dJ : 0.0;             // C12 C21
        elements[i].matriz[4] = dJ != 0 ? ((J2-J4)*-1*J2 + (J3-J1)*J1)*E.eps/dJ : 0.0;          // C13 C31
        elements[i].matriz[5] = dJ != 0 ? (J4*-1*J2 - J3*J1)*E.eps/dJ : 0.0;                    // C23 C32
    }

    // Calcula dsum e rsum.
    int n1, n2, n3;
    for (i = 0; i < ne; i++) {
        E = elements[i];
        n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

        dsum[n1] += E.matriz[0];
        dsum[n2] += E.matriz[1];
        dsum[n3] += E.matriz[2];

        rsum[n1] -= E.matriz[3]*V[n2] - E.matriz[4]*V[n3];
        rsum[n2] -= E.matriz[3]*V[n2] - E.matriz[5]*V[n3];
        rsum[n3] -= E.matriz[5]*V[n2] - E.matriz[4]*V[n1];
    }

    // Inicializa vetor de resíduos
    float ri, erri = 0;
    for (i = 0; i < nn; i++) {
        ri = nodes[i].calc ? rsum[i] - dsum[i]*V[i] : 0.0;
        r[i] = ri;
        if (ri != 0)
            erri += pow(ri, 2);
    }
    erri = sqrt(erri);

    // Iterações.
    int k = 1;
    float rho, rhop, alpha, beta, somaPQ, errf, errlat = 10*errmin;
    while(errlat > errmin && k < kmax) {
        // Pré-condicionador Jacobi e calcula Rho.
        rho = 0.0;
        for (i = 0; i < nn; i++) {
            z[i] = r[i]/dsum[i];
            rho += z[i]*r[i];
        }

        // Calcula P = Z + BETA*P
        if (k==1)
            for (i = 0; i < nn; i++)
                p[i] = z[i];
        else {
            beta = rho/rhop;
            for (i = 0; i < nn; i++)
                p[i] = z[i] + beta*p[i];
        }

        // Calcula Q = A*P
        for (i = 0; i < nn; i++)
            q[i] = 0.0;

        for (i = 0; i < ne; i++) {
            E = elements[i];
            n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

            q[n1] += E.matriz[0]*p[n1] + E.matriz[3]*p[n2] + E.matriz[4]*p[n3];
            q[n2] += E.matriz[3]*p[n1] + E.matriz[1]*p[n2] + E.matriz[5]*p[n3];
            q[n3] += E.matriz[4]*p[n1] + E.matriz[5]*p[n2] + E.matriz[2]*p[n3];
        }
        for (i = 0; i < nn; i++)
            if (!nodes[i].calc)
                q[i] = p[i];

        // Calcula Alpha
        somaPQ = 0.0;
        for (i = 0; i < nn; i++)
            somaPQ += p[i]*q[i];
        alpha = rho/somaPQ;

        // Atualiza 'x' e calcula o novo resíduo.
        errf = 0.0;
        for (i = 0; i < nn; i++) {
            V[i] += alpha*p[i];
            r[i] -= alpha*q[i];
            errf += pow(r[i], 2);
        }
        errf = sqrt(errf);
        errlat = errf/erri;

        rhop = rho;
        k++;
    }


    free(rsum);
    free(dsum);
    free(r);
    free(z);
    free(p);
    free(q);
    return k;
}
