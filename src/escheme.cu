#include "hip/hip_runtime.h"
/*
 The MIT License (MIT)

 Copyright (c) 2014 Leonardo Kewitz

 Permission is hereby granted, free of charge, to any person obtaining a copy
 of this software and associated documentation files (the "Software"), to deal
 in the Software without restriction, including without limitation the rights
 to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 copies of the Software, and to permit persons to whom the Software is
 furnished to do so, subject to the following conditions:

 The above copyright notice and this permission notice shall be included in all
 copies or substantial portions of the Software.

 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 SOFTWARE.
 */

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "./cuda_snippets.h"
#include "./escheme.h"


// Função externa que processa o problema no CPU.
//    ne: número de elementos.
//    nn: número de nós.
//    kmax: número máximo de iterações.
//    errmin: erro mínimo para considerar a convergência do resultado.
//    elements: array de elementos da malha.
//    nodes: array de nós da malha.
//    V: vetor de tensões dos nós.
//    verbose: se 'true' imprime informações do algorítmo.
//    bench: array de tempos de processamento para benchmarking.
extern "C" int runCPU(int ne, int nn, int kmax, float errmin,
                      elementri *elements, node *nodes, float *V, bool verbose,
                      float *bench) {
    int i;
    // clock_t t;

    float *rsum = static_cast<float*>(malloc(nn*sizeof(float)));
    float *dsum = static_cast<float*>(malloc(nn*sizeof(float)));
    float *r = static_cast<float*>(malloc(nn*sizeof(float)));
    float *z = static_cast<float*>(malloc(nn*sizeof(float)));
    float *p = static_cast<float*>(malloc(nn*sizeof(float)));
    float *q = static_cast<float*>(malloc(nn*sizeof(float)));
    // float *Vos = (float*) malloc(nn*sizeof(float));
    // memcpy(Vos, V, nn*sizeof(float));

    // Inicialização dos vetores.
    for (i = 0; i < nn; i++) {
        rsum[i] = 0.0;
        dsum[i] = 0.0;
    }

    // Pre-processamento. Calcula as matrizes de contribuição dos elementos.
    float J1, J2, J3, J4, dJ;
    elementri E;
    node N1, N2, N3;
    for (i = 0; i < ne; i++) {
        E = elements[i];
        N1 = nodes[E.nodes[0]]; N2 = nodes[E.nodes[1]]; N3 = nodes[E.nodes[2]];

        // Calcula argumentos necessários
        J1 = N2.x - N1.x;
        J2 = N2.y - N1.y;
        J3 = N3.x - N1.x;
        J4 = N3.y - N1.y;
        dJ = 2*(J1*J4 - J3*J2);

        // Calcula a matriz de contribuições do elemento.
        elements[i].matriz[0] = dJ != 0.0 ?
            (pow(J2-J4, 2) + pow(J3-J1, 2))*E.eps/dJ : 0.0;
        elements[i].matriz[1] = dJ != 0.0 ?
            (pow(J4, 2) + pow(J3, 2))*E.eps/dJ : 0.0;
        elements[i].matriz[2] = dJ != 0.0 ?
            (pow(J2, 2) + pow(J1, 2))*E.eps/dJ : 0.0;
        elements[i].matriz[3] = dJ != 0.0 ?
            ((J2-J4)*J4 - (J3-J1)*J3)*E.eps/dJ : 0.0;
        elements[i].matriz[4] = dJ != 0.0 ?
            ((J2-J4)*-1*J2 + (J3-J1)*J1)*E.eps/dJ : 0.0;
        elements[i].matriz[5] = dJ != 0.0 ?
            (J4*-1*J2 - J3*J1)*E.eps/dJ : 0.0;
    }

    // Calcula dsum e rsum.
    int n1, n2, n3;
    for (i = 0; i < ne; i++) {
        E = elements[i];
        n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

        dsum[n1] += E.matriz[0];
        dsum[n2] += E.matriz[1];
        dsum[n3] += E.matriz[2];

        rsum[n1] -= E.matriz[3]*V[n2] - E.matriz[4]*V[n3];
        rsum[n2] -= E.matriz[3]*V[n1] - E.matriz[5]*V[n3];
        rsum[n3] -= E.matriz[4]*V[n1] - E.matriz[5]*V[n2];
    }

    // Inicializa vetor de resíduos
    // r = b - Ax
    float ri, erri = 0.0;
    for (i = 0; i < nn; i++) {
        ri = nodes[i].calc ? rsum[i] - dsum[i]*V[i] : 0.0;
        r[i] = ri;
        if (ri != 0)
            erri += pow(ri, 2);
    }
    erri = sqrt(erri);

    // Iterações.
    int k = 1;
    float rho, rhop, alpha, beta, somaPQ, errf, errlat = 10*errmin;
    while (errlat > errmin && k < kmax) {
        rho = 0.0;
        // Pré-condicionador Jacobi e calcula Rho.
        for (i = 0; i < nn; i++) {
            z[i] = r[i]/dsum[i];
            rho += z[i]*r[i];
        }

        // Calcula P = Z + BETA*P
        if (k == 1) {
            for (i = 0; i < nn; i++)
                p[i] = z[i];
        } else {
            beta = rho/rhop;
            for (i = 0; i < nn; i++)
                p[i] = z[i] + beta*p[i];
        }

        // Calcula Q = A*P
        for (i = 0; i < nn; i++)
            q[i] = 0.0;
        for (i = 0; i < ne; i++) {
            E = elements[i];
            n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

            q[n1] += E.matriz[0]*p[n1] + E.matriz[3]*p[n2] + E.matriz[4]*p[n3];
            q[n2] += E.matriz[3]*p[n1] + E.matriz[1]*p[n2] + E.matriz[5]*p[n3];
            q[n3] += E.matriz[4]*p[n1] + E.matriz[5]*p[n2] + E.matriz[2]*p[n3];
        }

        for (i = 0; i < nn; i++)
            if (!nodes[i].calc)
                q[i] = p[i];

        // Calcula Alpha
        somaPQ = 0.0;
        for (i = 0; i < nn; i++)
            somaPQ += p[i]*q[i];
        alpha = rho/somaPQ;

        // Atualiza 'x' e calcula o novo resíduo.
        errf = 0.0;
        for (i = 0; i < nn; i++) {
            V[i] += alpha*p[i];
            r[i] -= alpha*q[i];
            errf += pow(r[i], 2);
        }
        errf = sqrt(errf);
        errlat = errf/erri;

        rhop = rho;
        k++;
    }


    free(rsum);
    free(dsum);
    free(r);
    free(z);
    free(p);
    free(q);
    return k;
}

extern "C" int runCPUCG(int ne, int nn, int kmax, float errmin,
                      elementri *elements, node *nodes, float *V, bool verbose,
                      float *bench) {
    int i, k;
    float rho, rho_, alpha, beta;
    float *r = static_cast<float*>(malloc(nn*sizeof(float)));
    float *d = static_cast<float*>(malloc(nn*sizeof(float)));
    float *q = static_cast<float*>(malloc(nn*sizeof(float)));
    float *rsum = static_cast<float*>(malloc(nn*sizeof(float)));
    float *dsum = static_cast<float*>(malloc(nn*sizeof(float)));

    // Inicialização dos vetores.
    for (i = 0; i < nn; i++) {
        rsum[i] = 0.0;
        dsum[i] = 0.0;
    }

    // Pre-processamento. Calcula as matrizes de contribuição dos elementos.
    float J1, J2, J3, J4, dJ;
    elementri E;
    node N1, N2, N3;
    for (i = 0; i < ne; i++) {
        E = elements[i];
        N1 = nodes[E.nodes[0]]; N2 = nodes[E.nodes[1]]; N3 = nodes[E.nodes[2]];

        // Calcula argumentos necessários
        J1 = N2.x - N1.x;
        J2 = N2.y - N1.y;
        J3 = N3.x - N1.x;
        J4 = N3.y - N1.y;
        dJ = 2*(J1*J4 - J3*J2);

        // Calcula a matriz de contribuições do elemento.
        elements[i].matriz[0] = dJ != 0.0 ?
            (pow(J2-J4, 2) + pow(J3-J1, 2))*E.eps/dJ : 0.0;
        elements[i].matriz[1] = dJ != 0.0 ?
            (pow(J4, 2) + pow(J3, 2))*E.eps/dJ : 0.0;
        elements[i].matriz[2] = dJ != 0.0 ?
            (pow(J2, 2) + pow(J1, 2))*E.eps/dJ : 0.0;
        elements[i].matriz[3] = dJ != 0.0 ?
            ((J2-J4)*J4 - (J3-J1)*J3)*E.eps/dJ : 0.0;
        elements[i].matriz[4] = dJ != 0.0 ?
            ((J2-J4)*-1*J2 + (J3-J1)*J1)*E.eps/dJ : 0.0;
        elements[i].matriz[5] = dJ != 0.0 ?
            (J4*-1*J2 - J3*J1)*E.eps/dJ : 0.0;
    }

    // Calcula dsum e rsum.
    int n1, n2, n3;
    for (i = 0; i < ne; i++) {
        E = elements[i];
        n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

        dsum[n1] += E.matriz[0];
        dsum[n2] += E.matriz[1];
        dsum[n3] += E.matriz[2];

        rsum[n1] -= E.matriz[3]*V[n2] - E.matriz[4]*V[n3];
        rsum[n2] -= E.matriz[3]*V[n1] - E.matriz[5]*V[n3];
        rsum[n3] -= E.matriz[4]*V[n1] - E.matriz[5]*V[n2];
    }

    // r = b - Ax
    rho_ = 0.0;
    float ri;
    for (i = 0; i < nn; i++) {
        if (nodes[i].calc) {
            ri = rsum[i] - dsum[i]*V[i];
            rho_ += pow(ri, 2);
        } else {
            ri = 0.0;
        }
        r[i] = ri;
        d[i] = ri;
    }
    rho = rho_;

    float dq;
    k = 1;
    errmin = pow(errmin, 2);
    while (k < kmax && rho_ > errmin*rho) {
        // q = Ad
        for (i = 0; i < nn; i++)
            q[i] = 0.0;
        for (i = 0; i < ne; i++) {
            E = elements[i];
            n1 = E.nodes[0]; n2 = E.nodes[1]; n3 = E.nodes[2];

            q[n1] += E.matriz[0]*d[n1] + E.matriz[3]*d[n2] + E.matriz[4]*d[n3];
            q[n2] += E.matriz[3]*d[n1] + E.matriz[1]*d[n2] + E.matriz[5]*d[n3];
            q[n3] += E.matriz[4]*d[n1] + E.matriz[5]*d[n2] + E.matriz[2]*d[n3];
        }

        // alpha = rho_/d'q
        dq = 0.0;
        for (i = 0; i < nn; i++)
            dq += d[i]*q[i];
        alpha = dq != 0 ? rho_/dq : 0.0;

        // x = x + alpha*d
        for (i = 0; i < nn; i++)
            if (nodes[i].calc)
                V[i] += alpha*d[i];

        rho = rho_;
        rho_ = 0.0;
        for (i = 0; i < nn; i++) {
            if (nodes[i].calc) {
                if (k%50 == 1)
                    ri = rsum[i] - dsum[i]*V[i];
                else
                    ri -= alpha*q[i];
                r[i] = ri;
                rho_ += pow(ri, 2);
            }
        }

        beta = rho != 0 ? rho_/rho : 0.0;
        for (i = 0; i < nn; i++)
            d[i] = r[i] + beta*d[i];

        k++;
    }

    free(dsum);
    free(rsum);
    free(r);
    free(d);
    free(q);

    return k;
}

extern "C" int testeCG(int n, int kmax, float err, float* A, float* x,
                       float* b) {
    int i, j, k = 1;
    float alpha, beta, sum1, sum2, sum3 = 1, sum4;
    float *r = (float*)malloc(n*sizeof(float));
    float *p = (float*)malloc(n*sizeof(float));
    float *u = (float*)malloc(n*sizeof(float));

    for (i = 0; i < n; i++) {
        p[i] = b[i];
        r[i] = b[i];
    }

    while (k < kmax || sum3 > err) {
        for (j = 0; j < n; j++) {
            u[j] = 0.0;
            for (i = 0; i < n; i++)
                u[j] += A[i*n + j]*p[i];
        }

        sum1 = 0.0; sum2 = 0.0;
        for (i = 0; i < n; i++) {
            sum1 += p[i]*r[i];
            sum2 += p[i]*u[i];
        }

        alpha = sum2 != 0.0 ? sum1/sum2 : 0.0;

        for (i = 0; i < n; i++) {
            x[i] += alpha*p[i];
            r[i] -= alpha*u[i];
        }

        sum3 = 0.0; sum4 = 0.0;
        for (i = 0; i < n; i++) {
            sum3 += r[i]*r[i];
            sum4 += r[i]*u[i];
        }

        beta = sum2 != 0.0 ? -sum4/sum2 : 0.0;

        for (i = 0; i < n; i++) {
            p[i] = r[i] + beta*p[i];
        }

        k++;
    }

    free(r);
    free(p);
    free(u);

    return k;
}

extern "C" int testeCG2(int n, int kmax, float err, float* A, float* x,
                        float* b) {
    int i, j, k = 1;
    float alpha, beta;
    float rho, rho_n, dq;
    float *r = (float*)malloc(n*sizeof(float));
    float *d = (float*)malloc(n*sizeof(float));
    float *q = (float*)malloc(n*sizeof(float));

    for (i = 0; i < n; i++) {
        r[i] = b[i];
        d[i] = b[i];
    }

    rho_n = 0;
    for (i = 0; i < n; i++)
        rho_n += pow(r[i], 2);
    rho = rho_n;

    while (k < kmax && fabs(sqrt(rho_n)) > err) {
        for (j = 0; j < n; j++) {
            q[j] = 0.0;
            for (i = 0; i < n; i++)
                q[j] += A[i*n + j]*d[i];
        }

        dq = 0;
        for (i = 0; i < n; i++)
            dq += d[i]*q[i];
        alpha = rho_n/dq;

        for (i = 0; i < n; i++) {
            x[i] = x[i] + alpha*d[i];
            r[i] = r[i] - alpha*q[i];
        }

        rho = rho_n;
        rho_n = 0;
        for (i = 0; i < n; i++)
            rho_n += pow(r[i], 2);

        beta = rho_n/rho;

        for (i = 0; i < n; i++) {
            d[i] = r[i] + beta*d[i];
        }

        k++;
    }

    free(r);
    free(d);
    free(q);

    return k;
}

extern "C" int testeSD(int n, int kmax, float err, float* A, float* x,
                        float* b) {
    int i, j, k = 1;
    float rho, rq, alpha;
    float *r = (float*)malloc(n*sizeof(float));
    float *q = (float*)malloc(n*sizeof(float));

    for (i = 0; i < n; i++)
        r[i] = b[i];

    rho = 0;
    for (i = 0; i < n; i++)
        rho += pow(r[i], 2);

    while (k < kmax && fabs(rho) > err) {
        for (j = 0; j < n; j++) {
            q[j] = 0.0;
            for (i = 0; i < n; i++)
                q[j] += A[i*n + j]*r[i];
        }

        rq = 0;
        for (i = 0; i < n; i++)
            rq += r[i]*q[i];

        alpha = rho/rq;

        for (i = 0; i < n; i++) {
            x[i] = x[i] + alpha*r[i];
            r[i] = r[i] - alpha*q[i];
        }

        printf("\n ");
        for (i = 0; i < n; i++) {
            printf("%.4f ", r[i]);
            rho += pow(r[i], 2);
        }

        k++;
    }

    return k;
}
