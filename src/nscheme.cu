#include "hip/hip_runtime.h"
/*
 The MIT License (MIT)

 Copyright (c) 2014 Leonardo Kewitz

 Permission is hereby granted, free of charge, to any person obtaining a copy
 of this software and associated documentation files (the "Software"), to deal
 in the Software without restriction, including without limitation the rights
 to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 copies of the Software, and to permit persons to whom the Software is
 furnished to do so, subject to the following conditions:

 The above copyright notice and this permission notice shall be included in all
 copies or substantial portions of the Software.

 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 SOFTWARE.
 */

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "./cuda_snippets.h"
#include "./nscheme.h"

#define DEBUG true
#define BSIZE 256

#define cma(a, b, c, d, e) CudaSafeCall(hipMemcpyAsync(a, b, c, d, e))


// Kernel responsável por uma iteração.
__global__ void kernel_node(int nn, float R, float errmin, elementri *elements,
    node *nodes, float *V, int *conv) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nn) return;

    node Node = nodes[i];

    int e, c;
    float diag_sum = 0.0, right_sum = 0.0, Vo = V[Node.i], Vi, diff;
    elementri Element;

    for (e = 0; e < Node.ne; e++) {
        Element = elements[Node.elements[e]];
        if (Node.i == Element.nodes[0]) {
            diag_sum  += Element.matriz[0];
            right_sum -= Element.matriz[3]*V[Element.nodes[1]];
            right_sum -= Element.matriz[4]*V[Element.nodes[2]];
        }
        if (Node.i == Element.nodes[1]) {
            diag_sum += Element.matriz[1];
            right_sum -= Element.matriz[3]*V[Element.nodes[0]];
            right_sum -= Element.matriz[5]*V[Element.nodes[2]];
        }
        if (Node.i == Element.nodes[2]) {
            diag_sum += Element.matriz[2];
            right_sum -= Element.matriz[4]*V[Element.nodes[0]];
            right_sum -= Element.matriz[5]*V[Element.nodes[1]];
        }
    }

    Vi = diag_sum == 0 ? 0.0f : fdividef(right_sum, diag_sum);
    diff = Vi - Vo;
    Vi += R*diff;
    c = fabs(diff) > errmin*fabs(Vi);
    atomicOr(conv, c);
    V[Node.i] = Vi;
}

// Kernel de pre-processamento responsável por calcular as matrizes de contribu-
// ição de todos os elementos.
__global__ void kernel_element(int ne, elementri *elements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    elementri *E = &elements[i];

    // Calcula argumentos necessários
    float J1, J2, J3, J4, dJ;
    J1 = E->x[1] - E->x[0];
    J2 = E->y[1] - E->y[0];
    J3 = E->x[2] - E->x[0];
    J4 = E->y[2] - E->y[0];
    dJ = 2*(J1*J4 - J3*J2);

    // Calcula a matriz de contribuições do elemento.
    E->matriz[0] = (pow(J2-J4, 2) + pow(J3-J1, 2))/dJ;   // C11
    E->matriz[1] = (pow(J4, 2) + pow(J3, 2))/dJ;         // C22
    E->matriz[2] = (pow(J2, 2) + pow(J1, 2))/dJ;         // C33
    E->matriz[3] = ((J2-J4)*J4 - (J3-J1)*J3)/dJ;       // C12 C21
    E->matriz[4] = ((J2-J4)*-1*J2 + (J3-J1)*J1)/dJ;    // C13 C31
    E->matriz[5] = (J4*-1*J2 - J3*J1)/dJ;              // C23 C32
}

// Calcula espaço teórico máximo de nós e elementos que cabem na memória da GPU.
extern "C" unsigned int alloc(const int nn) {
    hipDeviceProp_t prop = getInfo();
    unsigned int gm = prop.totalGlobalMem*.9 - sizeof(float)*nn;
    hipDeviceReset();
    return cast(unsigned int, gm / (sizeof(node) + 6*sizeof(elementri)));
}

// Função externa que processa o problema, responsável por alocar a memória no
// device e invocar todas os kernels necessários.
extern "C" int runGPU(int ng, int nn, int kmax, float R, float errmin,
    group *groups, float *V, bool verbose, float *bench) {
    // Inicia cronômetro do benchmark.
    clock_t t = clock();
    hipDeviceReset();
    // Aloca variáveis.
    int k = 1, g, conv, *d_conv;
    float *d_V;
    group G;
    elementri *d_elements;
    node *d_nodes;

    unsigned int maxn = alloc(nn);

    // Malloc e Memcpy de variáveis globais.
    smalloc(&d_V, sizeof(float)*nn);
    smalloc(&d_conv, sizeof(int));
    smalloc(&d_nodes, sizeof(node)*maxn);
    smalloc(&d_elements, sizeof(elementri)*maxn*6);
    smemcpy(d_V, V, sizeof(float)*nn, hipMemcpyHostToDevice);

    // Iterações
    conv = 1;
    while (conv == 1 && k < kmax) {
        conv = 0;
        smemcpy(d_conv, &conv, sizeof(int), hipMemcpyHostToDevice);
        for (g = 0; g < ng; g++) {
            hipDeviceSynchronize();
            G = groups[g];
            // Memcpy e processamento dos elementos.
            smemcpy(d_elements, G.elements, sizeof(elementri)*G.ne,
                hipMemcpyHostToDevice);
            kernel_element<<<(1 + G.ne/BSIZE), BSIZE>>>(G.ne, d_elements);
            // Memcpy dos nós enquanto se processa os elementos.
            smemcpy(d_nodes, G.nodes, sizeof(node)*G.nn,
                hipMemcpyHostToDevice);
            kernel_node<<<(1 + G.nn/BSIZE), BSIZE>>>(G.nn, R, errmin,
                d_elements, d_nodes, d_V, d_conv);
        }
        hipDeviceSynchronize();
        smemcpy(&conv, d_conv, sizeof(int), hipMemcpyDeviceToHost);
        k++;
    }

    smemcpy(V, d_V, sizeof(float)*nn, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_elements); hipFree(d_nodes);
    hipFree(d_V); hipFree(d_conv);

    t = clock() - t;
    bench[0] = cast(float, t)/CLOCKS_PER_SEC;
    return k;
}



extern "C" int streamGPU(int ng, int nn, int kmax, float R, float errmin,
    group *groups, float *V, bool verbose, float *bench) {
    // Inicia cronômetro do benchmark.
    clock_t t = clock();
    hipDeviceReset();
    // Aloca variáveis.
    int k = 1, g, conv, *d_conv;
    bool copye = true, copyn = true;
    float *d_V;
    elementri *d_elements, *pe;
    node *d_nodes, *pn;
    group *G;

    unsigned int maxn = alloc(nn);
    size_t sN = sizeof(node)*maxn;
    size_t sE = sizeof(elementri)*maxn*6;

    // Malloc e Memcpy de variáveis globais.
    smalloc(&d_V, sizeof(float)*nn);
    smalloc(&d_conv, sizeof(int));
    smalloc(&d_nodes, sN);
    smalloc(&d_elements, sE);
    // Inicia a cópia do vetor V.
    smemcpy(d_V, V, sizeof(float)*nn, hipMemcpyHostToDevice);

    // Cria streams.
    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i)
        hipStreamCreate(&stream[i]);

    // Define ponteiros temporários para streaming.
    pe = d_elements;
    pn = d_nodes;

    // Iterações
    conv = 1;
    while (conv == 1 && k < kmax) {
        conv = 0;
        smemcpy(d_conv, &conv, sizeof(int), hipMemcpyHostToDevice);

        for (g = 0; g < ng; g++) {
            G = &groups[g];
            if (copye) {
                pe = d_elements;
                cma(pe, G->elements, sizeof(elementri)*G->ne,
                    hipMemcpyHostToDevice, stream[0]);
            }
            if (copyn) {
                pn = d_nodes;
                cma(pn, G->nodes, sizeof(node)*G->nn,
                    hipMemcpyHostToDevice, stream[1]);
            }
            kernel_element<<<(1 + G->ne/BSIZE), BSIZE, 0, stream[0]>>>(G->ne,
                pe);
            hipDeviceSynchronize();

            kernel_node<<<(1 + G->nn/BSIZE), BSIZE, 0, stream[0]>>>(G->nn, R,
                errmin, pe, pn, d_V, d_conv);
            // Se não for o último grupo, já copia novos elementos.
            if (g < ng-1 && sE - sizeof(elementri)*G->ne > sizeof(elementri)*groups[g+1].ne) {
                pe += G->ne;
                cma(pe, groups[g+1].elements, sizeof(elementri)*groups[g+1].ne,
                    hipMemcpyHostToDevice, stream[1]);
                copye = false;
            } else {
                copye = true;
            }
            hipDeviceSynchronize();
        }
        hipDeviceSynchronize();
        smemcpy(&conv, d_conv, sizeof(int), hipMemcpyDeviceToHost);
        k++;
    }

    smemcpy(V, d_V, sizeof(float)*nn, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for (int i = 0; i < 2; ++i)
        hipStreamDestroy(stream[i]);

    hipFree(d_V); hipFree(d_conv);
    hipFree(d_elements); hipFree(d_nodes);

    t = clock() - t;
    bench[0] = cast(float, t)/CLOCKS_PER_SEC;

    return k;
}

void integ_element(elementri *E) {
    float J1, J2, J3, J4, dJ;
    J1 = E->x[1] - E->x[0];
    J2 = E->y[1] - E->y[0];
    J3 = E->x[2] - E->x[0];
    J4 = E->y[2] - E->y[0];
    dJ = 2*(J1*J4 - J3*J2);

    // Calcula a matriz de contribuições do elemento.
    E->matriz[0] = (pow(J2-J4, 2) + pow(J3-J1, 2))/dJ;   // C11
    E->matriz[1] = (pow(J4, 2) + pow(J3, 2))/dJ;         // C22
    E->matriz[2] = (pow(J2, 2) + pow(J1, 2))/dJ;         // C33
    E->matriz[3] = ((J2-J4)*J4 - (J3-J1)*J3)/dJ;         // C12 C21
    E->matriz[4] = ((J2-J4)*-1*J2 + (J3-J1)*J1)/dJ;      // C13 C31
    E->matriz[5] = (J4*-1*J2 - J3*J1)/dJ;                // C23 C32
}

void calc_node(node N, float errmin, float R, float *V, elementri *elements,
    bool *run) {
    int e;
    float diag_sum = 0.0f, right_sum = 0.0f, Vi, Vo, diff;
    elementri E;
    Vo = V[N.i];

    for (e = 0; e < N.ne; e++) {
        E = elements[N.elements[e]];
        if (N.i == E.nodes[0]) {
            diag_sum  += E.matriz[0];
            right_sum -= E.matriz[3]*V[E.nodes[1]];
            right_sum -= E.matriz[4]*V[E.nodes[2]];
        }
        if (N.i == E.nodes[1]) {
            diag_sum += E.matriz[1];
            right_sum -= E.matriz[3]*V[E.nodes[0]];
            right_sum -= E.matriz[5]*V[E.nodes[2]];
        }
        if (N.i == E.nodes[2]) {
            diag_sum += E.matriz[2];
            right_sum -= E.matriz[4]*V[E.nodes[0]];
            right_sum -= E.matriz[5]*V[E.nodes[1]];
        }
    }

    Vi = diag_sum == 0 ? 0.0 : right_sum/diag_sum;
    diff = Vi - Vo;
    Vi += R*diff;
    *run |= (fabs(diff) > errmin*fabs(Vi));
    V[N.i] = Vi;
}

extern "C" int runCPU(int ng, int nn, int kmax, float R, float errmin,
    group *groups, float *V, bool verbose, float *bench) {
    // Inicia cronômetro do benchmark.
    clock_t t = clock();
    // Aloca variáveis.
    int i, j, k = 1;

    // Loop principal das iterações.
    bool run = true;
    while (run && k < kmax) {
        run = false;
        // Loop de grupo emulado.
        for (i = 0; i < ng; i++) {
            group G = groups[i];
            // Integra os elementos do Grupo
            for (j = 0; j < G.ne; j++)
                integ_element(&G.elements[j]);
            // Calcula os potenciais nos nós do Grupo.
            for (j = 0; j < G.nn; j++)
                calc_node(G.nodes[j], errmin, R, V, G.elements, &run);
        }
        k++;
    }

    t = clock() - t;
    bench[0] = cast(float, t)/CLOCKS_PER_SEC;
    return k;
}

extern "C" void test_group(int ng, group *groups) {
    unsigned int i;

    for (i = 0; i < ng; i++) {
        group G = groups[i];
        printf("Group %i has %i nodes and %i elements.\n", i, G.nn, G.ne);
        printf("Nodes: %p\t Elements: %p\n\n", G.nodes, G.elements);
    }
}
