#include "hip/hip_runtime.h"
/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2014 Leonardo Kewitz
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 */

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_snippets.h"
#include "nscheme.h"

__global__ void kernel_iter(int nn, elementri *elements, node *nodes, double *V) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nn) return;

    int e;
    double diag_sum = 0.0, right_sum = 0.0, sn;

    node Node = nodes[i];
    if (Node.calc == false) return;
    sn = 0.0;

    elementri Element;
    for (e = 0; e < Node.ne; e++) {
        Element = elements[Node.elements[e]];
        right_sum += sn;
        if (Node.i == Element.nodes[0]) {
            diag_sum  += Element.matriz[0];                     // A11
            right_sum -= Element.matriz[3]*V[Element.nodes[1]]; // A12
            right_sum -= Element.matriz[4]*V[Element.nodes[2]]; // A13
        }
        if (Node.i == Element.nodes[1]) {
            diag_sum += Element.matriz[1];                      // A22
            right_sum -= Element.matriz[3]*V[Element.nodes[1]]; // A21
            right_sum -= Element.matriz[5]*V[Element.nodes[2]]; // A23
        }
        if (Node.i == Element.nodes[2]) {
            diag_sum += Element.matriz[2];                      // A33
            right_sum -= Element.matriz[4]*V[Element.nodes[0]]; // A31
            right_sum -= Element.matriz[5]*V[Element.nodes[1]]; // A32
        }
    }

    V[Node.i] = diag_sum == 0 ? 0.0 : right_sum/diag_sum;

    return;
}

__global__ void kernel_pre(int ne, elementri *elements, node *nodes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    elementri E = elements[i];
    node N1 = nodes[E.nodes[0]], N2 = nodes[E.nodes[1]], N3 = nodes[E.nodes[2]];

    // Calcula argumentos necessários
    double J1, J2, J3, J4, dJ;
    J1 = (double)N2.x - (double)N1.x;
    J2 = (double)N2.y - (double)N1.y;
    J3 = (double)N3.x - (double)N1.x;
    J4 = (double)N3.y - (double)N1.y;
    dJ = 2*(J1*J4 - J3*J2);

    // Calcula a matriz de contribuições do elemento.
    elements[i].matriz[0] = (pow(J2-J4,2) + pow(J3-J1,2))/dJ;   // C11
    elements[i].matriz[1] = (pow(J4,2) + pow(J3,2))/dJ;         // C22
    elements[i].matriz[2] = (pow(J2,2) + pow(J1,2))/dJ;         // C33
    elements[i].matriz[3] = ((J2-J4)*J4 - (J3-J1)*J3)/dJ;       // C12 C21
    elements[i].matriz[4] = ((J2-J4)*-1*J2 + (J3-J1)*J1)/dJ;    // C13 C31
    elements[i].matriz[5] = (J4*-1*J2 - J3*J1)/dJ;              // C23 C32

    return;
}

extern "C" void run(int ne, int nn, int ks, elementri *elements, node *nodes, double *V) {
    hipDeviceProp_t prop;
    CudaSafeCall(hipGetDeviceProperties(&prop, 0) );
    printf("[!] Device Name: %s\n", prop.name);
    printf("[!] %s compiled in %s %s\n", __FILE__, __DATE__, __TIME__);

    int k;
    double *d_V;
    node *d_nodes;
    elementri *d_elements;
    const dim3 threads(512);
    const dim3 preblocks(1 + ne/512);
    const dim3 iterblocks(1 + nn/512);
    size_t s_Elements = sizeof(elementri)*ne,
           s_Nodes = sizeof(node)*nn,
           s_V = sizeof(double)*nn;

    // Malloc
    CudaSafeCall(hipMalloc(&d_elements, s_Elements));
    CudaSafeCall(hipMalloc(&d_nodes, s_Nodes));
    CudaSafeCall(hipMalloc(&d_V, s_V));
    // Memcpy
    CudaSafeCall(hipMemcpy(d_elements, elements, s_Elements, hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_nodes, nodes, s_Nodes, hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_V, V, s_V, hipMemcpyHostToDevice));

    // Pre-processamento
    kernel_pre<<<preblocks, threads>>>(ne, d_elements, d_nodes);
    hipDeviceSynchronize();

    // Iterações
    for (k = 0; k < ks; k++) {
        kernel_iter<<<iterblocks, threads>>>(nn, d_elements, d_nodes, d_V);
        hipDeviceSynchronize();
    }

    CudaSafeCall(hipMemcpy(V, d_V, s_V, hipMemcpyDeviceToHost));

    hipFree(d_V);
    hipFree(d_nodes);
    hipFree(d_elements);

    return;
}

extern "C" void teste_Arrays(int ne, int nn, elementri *elements, node *nodes) {
    int i, k;
    printf("\nStarting Node Test...\n\n");
    for (i = 0; i < nn; i++) {
        if (i%100 == 0) {
            printf("\tNode %i (%.3f, %.3f):\n", i, nodes[i].x, nodes[i].y);
            printf("\t\tElements:");
            for (k = 0; k < nodes[i].ne; k++) {
                printf(" %i", (int)nodes[i].elements[k]);
            }
            printf("\n");
        }
    }

    printf("\nStarting Elements Test...\n\n");
    for (i = 0; i < nn; i++) {
        if (i%100 == 0) {
            printf("\tElement %i:\n", i);
            printf("\t\tNodes:");
            for (k = 0; k < 3; k++) {
                printf(" %i", elements[i].nodes[k]);
            }
            printf("\n\t\tMatriz:");
            printf("\n\t\t\t%.3f %.3f %.3f",
                   elements[i].matriz[0],
                   elements[i].matriz[3],
                   elements[i].matriz[4]);
            printf("\n\t\t\t%.3f %.3f %.3f",
                   elements[i].matriz[3],
                   elements[i].matriz[1],
                   elements[i].matriz[5]);
            printf("\n\t\t\t%.3f %.3f %.3f",
                   elements[i].matriz[4],
                   elements[i].matriz[5],
                   elements[i].matriz[2]);
            printf("\n");
        }
    }
}
