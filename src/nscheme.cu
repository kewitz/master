#include "hip/hip_runtime.h"
/*
 The MIT License (MIT)

 Copyright (c) 2014 Leonardo Kewitz

 Permission is hereby granted, free of charge, to any person obtaining a copy
 of this software and associated documentation files (the "Software"), to deal
 in the Software without restriction, including without limitation the rights
 to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 copies of the Software, and to permit persons to whom the Software is
 furnished to do so, subject to the following conditions:

 The above copyright notice and this permission notice shall be included in all
 copies or substantial portions of the Software.

 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 SOFTWARE.
 */

#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "./cuda_snippets.h"
#include "./nscheme.h"

// Constantes
#define DEBUG true
#define STREAMED false
#define BSIZE 64
// Macros
#define cma(a, b, c, d, e) CudaSafeCall(hipMemcpyAsync(a, b, c, d, e))

// vec[i] = 0.0f
__global__ void kernel_util_zero(int nn, node *nodes, float *vec) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nn) return;
    vec[nodes[i].i] = 0.0f;
}

// Kernel de pre-processamento responsável por calcular as matrizes de contribu-
// ição de todos os elementos.
__global__ void kernel_element(int ne, element *elements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= ne) return;

    element E = elements[i];

    // Calcula gradN
    float q1 = E.y[1]-E.y[2], q2 = E.y[2]-E.y[0], q3 = E.y[0]-E.y[1];
    float r1 = E.x[2]-E.x[1], r2 = E.x[0]-E.x[2], r3 = E.x[1]-E.x[0];
    // Calcula det(gradN)
    float det = E.x[1]*E.y[2] + E.x[0]*E.y[1] + E.x[2]*E.y[0]
              - E.x[0]*E.y[2] - E.x[2]*E.y[1] - E.x[1]*E.y[0];
    float cof = (E.mat/det)/2;
    // Calcula a matriz de contribuições do elemento.
    elements[i].matriz[0] = (q1*q1 + r1*r1)*cof;
    elements[i].matriz[1] = (q2*q2 + r2*r2)*cof;
    elements[i].matriz[2] = (q3*q3 + r3*r3)*cof;
    elements[i].matriz[3] = (q1*q2 + r1*r2)*cof;
    elements[i].matriz[4] = (q1*q3 + r1*r3)*cof;
    elements[i].matriz[5] = (q2*q3 + r2*r3)*cof;
}

// Kernel responsável por uma iteração.
__global__ void kernel_node(int nn, float errmin, float R, element *elements,
    node *nodes, float *V, int *conv) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nn) return;

    node N = nodes[i];

    int e, c;
    float diag_sum = 0.0f, right_sum = 0.0f, Vo = V[N.i], Vi, diff;
    element E;

    for (e = 0; e < N.ne; e++) {
        E = elements[N.elements[e]];
        if (N.i == E.nodes[0]) {
            diag_sum  += E.matriz[0];
            right_sum -= E.matriz[3]*V[E.nodes[1]];
            right_sum -= E.matriz[4]*V[E.nodes[2]];
        } else if (N.i == E.nodes[1]) {
            diag_sum += E.matriz[1];
            right_sum -= E.matriz[3]*V[E.nodes[0]];
            right_sum -= E.matriz[5]*V[E.nodes[2]];
        } else {
            diag_sum += E.matriz[2];
            right_sum -= E.matriz[4]*V[E.nodes[0]];
            right_sum -= E.matriz[5]*V[E.nodes[1]];
        }
    }

    Vi = right_sum/diag_sum;
    diff = Vi - Vo;
    Vi += R*diff;
    c = (abs(diff/Vi) > errmin);
    atomicOr(conv, c);
    V[N.i] = Vi;
}

// Calcula espaço teórico máximo de nós e elementos que cabem na memória da GPU.
extern "C" unsigned int alloc(const int nn) {
    hipDeviceProp_t prop = getInfo();
    unsigned int gm = prop.totalGlobalMem*.9 - sizeof(float)*nn*2;
    hipDeviceReset();
    return cast(unsigned int, gm / (sizeof(node) + 6*sizeof(element)));
}

// Função externa que processa o problema, responsável por alocar a memória no
// device e invocar todas os kernels necessários.
extern "C" int runGPU(int ng, int nn, int kmax, float R, float errmin,
    group *groups, float *V, bool verbose, float *bench) {
    // Inicia cronômetro do benchmark.
    unsigned int maxn = alloc(nn);
    hipDeviceReset();
    clock_t t = clock();
    // Aloca variáveis.
    int k = 1, g, conv, *d_conv;
    float *d_V;
    group G;
    element *d_elements;
    node *d_nodes;

    // Malloc e Memcpy de variáveis globais.
    smalloc(&d_V, sizeof(float)*nn);
    smalloc(&d_conv, sizeof(int));
    smalloc(&d_nodes, sizeof(node)*maxn);
    smalloc(&d_elements, sizeof(element)*maxn*6);
    smemcpy(d_V, V, sizeof(float)*nn, hipMemcpyHostToDevice);

    // Iterações
    conv = 1;
    while (conv == 1 && k < kmax) {
        if (k%10)
            conv = 0;
        smemcpy(d_conv, &conv, sizeof(int), hipMemcpyHostToDevice);
        for (g = 0; g < ng; g++) {
            hipDeviceSynchronize();
            G = groups[g];
            if (ng > 1 || k == 1) {
                smemcpy(d_nodes, G.nodes, sizeof(node)*G.nn,
                    hipMemcpyHostToDevice);
                // Memcpy e processamento dos elementos.
                smemcpy(d_elements, G.elements, sizeof(element)*G.ne,
                    hipMemcpyHostToDevice);
            }
            hipDeviceSynchronize();
            kernel_element<<<(1 + G.ne/BSIZE), BSIZE>>>(G.ne, d_elements);
            // Memcpy dos nós enquanto se processa os elementos.
            hipDeviceSynchronize();
            kernel_node<<<(1 + G.nn/BSIZE), BSIZE>>>(G.nn, errmin, R,
                d_elements, d_nodes, d_V, d_conv);
        }
        hipDeviceSynchronize();
        if (k%10)
            smemcpy(&conv, d_conv, sizeof(int), hipMemcpyDeviceToHost);
        k++;
    }

    smemcpy(V, d_V, sizeof(float)*nn, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_elements); hipFree(d_nodes);
    hipFree(d_V); hipFree(d_conv);

    t = clock() - t;
    bench[0] = cast(float, t)/CLOCKS_PER_SEC;
    return k;
}

extern "C" int runGPUStream(int ng, int nn, int kmax, float R, float errmin,
    group *groups, float *V, bool verbose, float *bench) {
    // Inicia cronômetro do benchmark.
    unsigned int maxn = alloc(nn);
    hipDeviceReset();
    clock_t t = clock();
    // Aloca variáveis.
    int k = 1, g, conv, *d_conv;
    float *d_V;
    group *G, *Gn;
    element *d_elements;
    node *d_nodes;

    // Malloc e Memcpy de variáveis globais.
    smalloc(&d_V, sizeof(float)*nn);
    smalloc(&d_conv, sizeof(int));
    smalloc(&d_nodes, sizeof(node)*maxn);
    smalloc(&d_elements, sizeof(element)*maxn*6);
    smemcpy(d_V, V, sizeof(float)*nn, hipMemcpyHostToDevice);

    // Cria streams.
    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i)
        hipStreamCreate(&stream[i]);

    // Iterações
    conv = 1;
    while (conv == 1 && k < kmax) {
        conv = 0;
        smemcpy(d_conv, &conv, sizeof(int), hipMemcpyHostToDevice);
        G = &groups[0];
        G->d_elements = d_elements;
        G->d_nodes = d_nodes;
        cma(G->d_elements, G->elements, sizeof(element)*G->ne,
            hipMemcpyHostToDevice, stream[0]);
        cma(G->d_nodes, G->nodes, sizeof(node)*G->nn,
            hipMemcpyHostToDevice, stream[1]);
        for (g = 0; g < ng; g++) {
            G = &groups[g];
            // hipStreamSynchronize(stream[0]);
            kernel_element<<<(1 + G->ne/BSIZE), BSIZE, 0, stream[0]>>>(G->ne,
                G->d_elements);
            hipDeviceSynchronize();
            kernel_node<<<(1 + G->nn/BSIZE), BSIZE, 0, stream[0]>>>(G->nn,
                errmin, R, G->d_elements, G->d_nodes, d_V, d_conv);
            if (g < ng-1) {
                Gn = &groups[g+1];
                Gn->d_elements = G->d_elements+G->ne;
                Gn->d_nodes = G->d_nodes+G->nn;
                cma(Gn->d_elements, Gn->elements, sizeof(element)*Gn->ne,
                    hipMemcpyHostToDevice, stream[1]);
                cma(Gn->d_nodes, Gn->nodes, sizeof(node)*Gn->nn,
                    hipMemcpyHostToDevice, stream[1]);
            }
            hipDeviceSynchronize();
        }
        hipDeviceSynchronize();
        smemcpy(&conv, d_conv, sizeof(int), hipMemcpyDeviceToHost);
        k++;
    }

    smemcpy(V, d_V, sizeof(float)*nn, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for (int i = 0; i < 2; ++i)
        hipStreamDestroy(stream[i]);

    hipFree(d_elements); hipFree(d_nodes);
    hipFree(d_V); hipFree(d_conv);

    t = clock() - t;
    bench[0] = cast(float, t)/CLOCKS_PER_SEC;
    return k;
}

void integ_element(element *E) {
    float mat = E->mat;
    // Calcula gradN
    float q1 = E->y[1]-E->y[2], q2 = E->y[2]-E->y[0], q3 = E->y[0]-E->y[1];
    float r1 = E->x[2]-E->x[1], r2 = E->x[0]-E->x[2], r3 = E->x[1]-E->x[0];
    // Calcula det(gradN)
    float det = E->x[1]*E->y[2] + E->x[0]*E->y[1] + E->x[2]*E->y[0]
              - E->x[0]*E->y[2] - E->x[2]*E->y[1] - E->x[1]*E->y[0];
    float cof = (mat/det)/2.0;
    // Calcula a matriz de contribuições do elemento.
    E->matriz[0] = (q1*q1 + r1*r1)*cof;
    E->matriz[1] = (q2*q2 + r2*r2)*cof;
    E->matriz[2] = (q3*q3 + r3*r3)*cof;
    E->matriz[3] = (q1*q2 + r1*r2)*cof;
    E->matriz[4] = (q1*q3 + r1*r3)*cof;
    E->matriz[5] = (q2*q3 + r2*r3)*cof;
}

void calc_node(node N, float errmin, float R, float *V, element *elements,
    bool *run) {
    int e;
    float diag_sum = 0.0f, right_sum = 0.0f, Vo = V[N.i], Vi, diff;
    element E;

    for (e = 0; e < N.ne; e++) {
        E = elements[N.elements[e]];
        if (N.i == E.nodes[0]) {
            diag_sum  += E.matriz[0];
            right_sum -= E.matriz[3]*V[E.nodes[1]];
            right_sum -= E.matriz[4]*V[E.nodes[2]];
        }
        if (N.i == E.nodes[1]) {
            diag_sum += E.matriz[1];
            right_sum -= E.matriz[3]*V[E.nodes[0]];
            right_sum -= E.matriz[5]*V[E.nodes[2]];
        }
        if (N.i == E.nodes[2]) {
            diag_sum += E.matriz[2];
            right_sum -= E.matriz[4]*V[E.nodes[0]];
            right_sum -= E.matriz[5]*V[E.nodes[1]];
        }
    }

    Vi = right_sum/diag_sum;
    diff = Vi - Vo;
    Vi += R*diff;
    *run |= (fabs(diff/Vi) > errmin);
    V[N.i] = Vi;
}

extern "C" int runCPU(int ng, int nn, int kmax, float R, float errmin,
    group *groups, float *V, bool verbose, float *bench) {
    // Inicia cronômetro do benchmark.
    clock_t t = clock();
    // Aloca variáveis.
    int i, j, k = 1;

    // Loop principal das iterações.
    bool run = true;
    while (run && k < kmax) {
        run = false;
        // Loop de grupo emulado.
        for (i = 0; i < ng; i++) {
            group G = groups[i];
            // Integra os elementos do Grupo.
            for (j = 0; j < G.ne; j++)
                integ_element(&G.elements[j]);
            // Calcula os potenciais nos nós do Grupo.
            for (j = 0; j < G.nn; j++)
                calc_node(G.nodes[j], errmin, R, V, G.elements, &run);
        }
        k++;
    }

    t = clock() - t;
    bench[0] = cast(float, t)/CLOCKS_PER_SEC;
    return k;
}

extern "C" void test_group(int ng, group *groups) {
    unsigned int i;

    for (i = 0; i < ng; i++) {
        group G = groups[i];
        printf("Group %i has %i nodes and %i elements.\n", i, G.nn, G.ne);
        printf("Nodes: %p\t Elements: %p\n\n", G.nodes, G.elements);
    }
}
